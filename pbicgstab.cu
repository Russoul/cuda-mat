#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <typeinfo> // for usage of C++ typeid
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <conio.h>
#include <sstream>
#include <iostream>

#include "hipblas.h"
#include "hipsparse.h"
#include "helper_cusolver.h"
#include "mmio.h"

#include "mmio_wrapper.h"

#include "hip/hip_runtime_api.h"
#include "pbicgstab.h"


#include <>

__global__ void mult_spec(int n, double *a, double*b, double k, double *c){
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if(i >= n) return;

    c[i] = a[i] * b[i] * k;
}


static void gpu_pbicgstab(hipblasHandle_t cublasHandle, hipsparseHandle_t cusparseHandle, int n, int nnz,
                          const hipsparseMatDescr_t descra, /* the coefficient matrix in CSR format */
                          double *a, int *ia, int *ja,
                          const hipsparseMatDescr_t descrm, /* the preconditioner in CSR format, lower & upper triangular factor */
                          double *vm, int *im, int *jm,
                          cusparseSolveAnalysisInfo_t info_l, cusparseSolveAnalysisInfo_t info_u, /* the analysis of the lower and upper triangular parts */
                          double *f, double *r, double *rw, double *p, double *pw, double *s, double *t, double *v, double *x,
                          int maxit, double tol, bool debug)
{
    double rho, rhop, beta, alpha, negalpha, omega, negomega, temp, temp2;
    double nrmr, nrmr0;
    rho = 0.0;
    double zero = 0.0;
    double one  = 1.0;
    double mone = -1.0;
    int i=0;

    //WARNING: Analysis is done outside of the function (and the time taken by it is passed to the function in variable ttt_sv)

    //compute initial residual r0=b-Ax0 (using initial guess in x)


    checkCudaErrors(hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, descra, a, ia, ja, x, &zero, r));

    checkCudaErrors(hipblasDscal(cublasHandle, n, &mone, r, 1));
    checkCudaErrors(hipblasDaxpy(cublasHandle, n, &one, f, 1, r, 1));
    //copy residual r into r^{\hat} and p
    checkCudaErrors(hipblasDcopy(cublasHandle, n, r, 1, rw, 1));
    checkCudaErrors(hipblasDcopy(cublasHandle, n, r, 1, p, 1));
    checkCudaErrors(hipblasDnrm2(cublasHandle, n, r, 1, &nrmr0));

    if(debug)
        printf("gpu, init residual:norm %20.16f\n",nrmr0);

    for (i=0; i<maxit; ){
        rhop = rho;
        checkCudaErrors(hipblasDdot(cublasHandle, n, rw, 1, r, 1, &rho));

        if (i > 0){
            beta= (rho/rhop) * (alpha/omega);
            negomega = -omega;
            checkCudaErrors(hipblasDaxpy(cublasHandle,n, &negomega, v, 1, p, 1));
            checkCudaErrors(hipblasDscal(cublasHandle,n, &beta, p, 1));
            checkCudaErrors(hipblasDaxpy(cublasHandle,n, &one, r, 1, p, 1));
        }
            //preconditioning step (lower and upper triangular solve)

        checkCudaErrors(hipsparseSetMatFillMode(descrm,HIPSPARSE_FILL_MODE_LOWER));
        checkCudaErrors(hipsparseSetMatDiagType(descrm,HIPSPARSE_DIAG_TYPE_UNIT));
        checkCudaErrors(cusparseDcsrsv_solve(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,n,&one,descrm,vm,im,jm,info_l,p,t));

        checkCudaErrors(hipsparseSetMatFillMode(descrm,HIPSPARSE_FILL_MODE_UPPER));
        checkCudaErrors(hipsparseSetMatDiagType(descrm,HIPSPARSE_DIAG_TYPE_NON_UNIT));
        checkCudaErrors(cusparseDcsrsv_solve(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,n,&one,descrm,vm,im,jm,info_u,t,pw));


        //matrix-vector multiplication


        checkCudaErrors(hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, descra, a, ia, ja, pw, &zero, v));

        checkCudaErrors(hipblasDdot(cublasHandle,n, rw, 1, v, 1,&temp));
        alpha= rho / temp;
        negalpha = -(alpha);
        checkCudaErrors(hipblasDaxpy(cublasHandle,n, &negalpha, v, 1, r, 1));
        checkCudaErrors(hipblasDaxpy(cublasHandle,n, &alpha,        pw, 1, x, 1));
        checkCudaErrors(hipblasDnrm2(cublasHandle, n, r, 1, &nrmr));

        if(debug)
            std::cout << "i = " << i << ", residual norm (before precond) = " << nrmr << std::endl;

        if (nrmr < tol*nrmr0){
            break;
        }

        //preconditioning step (lower and upper triangular solve)
        checkCudaErrors(hipsparseSetMatFillMode(descrm,HIPSPARSE_FILL_MODE_LOWER));
        checkCudaErrors(hipsparseSetMatDiagType(descrm,HIPSPARSE_DIAG_TYPE_UNIT));
        checkCudaErrors(cusparseDcsrsv_solve(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,n, &one,descrm,vm,im,jm,info_l,r,t));

        checkCudaErrors(hipsparseSetMatFillMode(descrm,HIPSPARSE_FILL_MODE_UPPER));
        checkCudaErrors(hipsparseSetMatDiagType(descrm,HIPSPARSE_DIAG_TYPE_NON_UNIT));
        checkCudaErrors(cusparseDcsrsv_solve(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,n, &one,descrm,vm,im,jm,info_u,t,s));

        //matrix-vector multiplication


        checkCudaErrors(hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, descra, a, ia, ja, s, &zero, t));


        checkCudaErrors(hipblasDdot(cublasHandle,n, t, 1, r, 1,&temp));
        checkCudaErrors(hipblasDdot(cublasHandle,n, t, 1, t, 1,&temp2));
        omega= temp / temp2;
        negomega = -(omega);
        checkCudaErrors(hipblasDaxpy(cublasHandle,n, &omega, s, 1, x, 1));
        checkCudaErrors(hipblasDaxpy(cublasHandle,n, &negomega, t, 1, r, 1));

        checkCudaErrors(hipblasDnrm2(cublasHandle,n, r, 1,&nrmr));

        if(debug)
            std::cout << "i = " << i << ", residual norm = " << nrmr << std::endl;

        if (nrmr < tol*nrmr0){
            i++;
            break;
        }
        i++;
    }

}


bool bicgstab_lu_precond(int matrixN, int nnz, double* Aval, int* ArowsIndex, int* AcolsIndex, double* b,
                  int maxit, double tol, bool debug, double *res, double *dtAlg){
    hipblasHandle_t cublasHandle  = 0;
    hipsparseHandle_t cusparseHandle  = 0;
    hipsparseMatDescr_t descra= 0;
    hipsparseMatDescr_t descrm= 0;
    cusparseSolveAnalysisInfo_t info_l = 0;
    cusparseSolveAnalysisInfo_t info_u = 0;
    hipsparseStatus_t status1, status2;
    int matrixM = matrixN;
    double *devPtrAval       = 0;
    int    *devPtrAcolsIndex = 0;
    int    *devPtrArowsIndex = 0;
    double *devPtrMval       = 0;
    int    *devPtrMcolsIndex = 0;
    int    *devPtrMrowsIndex = 0;
    double *devPtrX = 0;
    double *devPtrF = 0;
    double *devPtrR = 0;
    double *devPtrRW= 0;
    double *devPtrP = 0;
    double *devPtrPW= 0;
    double *devPtrS = 0;
    double *devPtrT = 0;
    double *devPtrV = 0;
    double *x  = 0;
    double *f  = b;
    double *r  = 0;
    double *rw = 0;
    double *p  = 0;
    double *pw = 0;
    double *s  = 0;
    double *t  = 0;
    double *v  = 0;
    int matrixSizeAval, matrixSizeAcolsIndex, matrixSizeArowsIndex, mSizeAval;
    int arraySizeX, arraySizeF, arraySizeR, arraySizeRW, arraySizeP,  arraySizePW, arraySizeS, arraySizeT, arraySizeV, mNNZ;
    int base;




    matrixSizeAval       = nnz;
    matrixSizeAcolsIndex = matrixSizeAval;
    matrixSizeArowsIndex = matrixM + 1;
    base                 = ArowsIndex[0];

    if(debug)
        printf( "N=%d, nnz=%d\n", matrixN, nnz);


    /* set some extra parameters for lower triangular factor */
    mNNZ            = ArowsIndex[matrixM]-ArowsIndex[0];
    mSizeAval       = mNNZ;

    /* compressed sparse row */
    arraySizeX = matrixN;
    arraySizeF = matrixM;
    arraySizeR = matrixM;
    arraySizeRW= matrixM;
    arraySizeP = matrixN;
    arraySizePW= matrixN;
    arraySizeS = matrixM;
    arraySizeT = matrixM;
    arraySizeV = matrixM;

    /* initialize cublas */
    if (hipblasCreate(&cublasHandle) != HIPBLAS_STATUS_SUCCESS) {
        fprintf( stderr, "!!!! CUBLAS initialization error\n" );
        return false;
    }
    /* initialize cusparse */
    status1 = hipsparseCreate(&cusparseHandle);
    if (status1 != HIPSPARSE_STATUS_SUCCESS) {
        fprintf( stderr, "!!!! CUSPARSE initialization error\n" );
        return false;
    }
    /* create three matrix descriptors */
    status1 = hipsparseCreateMatDescr(&descra);
    status2 = hipsparseCreateMatDescr(&descrm);
    if ((status1 != HIPSPARSE_STATUS_SUCCESS) ||
        (status2 != HIPSPARSE_STATUS_SUCCESS)){
        fprintf( stderr, "!!!! CUSPARSE hipsparseCreateMatDescr (coefficient matrix or preconditioner) error\n" );
        return false;
    }

    /* allocate device memory for csr matrix and vectors */
    checkCudaErrors(hipMalloc ((void**)&devPtrX, sizeof(devPtrX[0]) * arraySizeX));
    checkCudaErrors(hipMalloc ((void**)&devPtrF, sizeof(devPtrF[0]) * arraySizeF));
    checkCudaErrors(hipMalloc ((void**)&devPtrR, sizeof(devPtrR[0]) * arraySizeR));
    checkCudaErrors(hipMalloc ((void**)&devPtrRW,sizeof(devPtrRW[0])* arraySizeRW));
    checkCudaErrors(hipMalloc ((void**)&devPtrP, sizeof(devPtrP[0]) * arraySizeP));
    checkCudaErrors(hipMalloc ((void**)&devPtrPW,sizeof(devPtrPW[0])* arraySizePW));
    checkCudaErrors(hipMalloc ((void**)&devPtrS, sizeof(devPtrS[0]) * arraySizeS));
    checkCudaErrors(hipMalloc ((void**)&devPtrT, sizeof(devPtrT[0]) * arraySizeT));
    checkCudaErrors(hipMalloc ((void**)&devPtrV, sizeof(devPtrV[0]) * arraySizeV));
    checkCudaErrors(hipMalloc ((void**)&devPtrAval, sizeof(devPtrAval[0]) * matrixSizeAval));
    checkCudaErrors(hipMalloc ((void**)&devPtrAcolsIndex, sizeof(devPtrAcolsIndex[0]) * matrixSizeAcolsIndex));
    checkCudaErrors(hipMalloc ((void**)&devPtrArowsIndex, sizeof(devPtrArowsIndex[0]) * matrixSizeArowsIndex));
    checkCudaErrors(hipMalloc ((void**)&devPtrMval, sizeof(devPtrMval[0]) * mSizeAval));

    /* allocate host memory for  vectors */
    x  = (double *)malloc (arraySizeX * sizeof(x[0]));
    //f  = (double *)malloc (arraySizeF * sizeof(f[0]));
    r  = (double *)malloc (arraySizeR * sizeof(r[0]));
    rw = (double *)malloc (arraySizeRW* sizeof(rw[0]));
    p  = (double *)malloc (arraySizeP * sizeof(p[0]));
    pw = (double *)malloc (arraySizePW* sizeof(pw[0]));
    s  = (double *)malloc (arraySizeS * sizeof(s[0]));
    t  = (double *)malloc (arraySizeT * sizeof(t[0]));
    v  = (double *)malloc (arraySizeV * sizeof(v[0]));


    /* clean memory */
    checkCudaErrors(hipMemset((void *)devPtrX,         0, sizeof(devPtrX[0])          * arraySizeX));
    checkCudaErrors(hipMemset((void *)devPtrF,         0, sizeof(devPtrF[0])          * arraySizeF));
    checkCudaErrors(hipMemset((void *)devPtrR,         0, sizeof(devPtrR[0])          * arraySizeR));
    checkCudaErrors(hipMemset((void *)devPtrRW,        0, sizeof(devPtrRW[0])         * arraySizeRW));
    checkCudaErrors(hipMemset((void *)devPtrP,         0, sizeof(devPtrP[0])          * arraySizeP));
    checkCudaErrors(hipMemset((void *)devPtrPW,        0, sizeof(devPtrPW[0])         * arraySizePW));
    checkCudaErrors(hipMemset((void *)devPtrS,         0, sizeof(devPtrS[0])          * arraySizeS));
    checkCudaErrors(hipMemset((void *)devPtrT,         0, sizeof(devPtrT[0])          * arraySizeT));
    checkCudaErrors(hipMemset((void *)devPtrV,         0, sizeof(devPtrV[0])          * arraySizeV));
    checkCudaErrors(hipMemset((void *)devPtrAval,      0, sizeof(devPtrAval[0])       * matrixSizeAval));
    checkCudaErrors(hipMemset((void *)devPtrAcolsIndex,0, sizeof(devPtrAcolsIndex[0]) * matrixSizeAcolsIndex));
    checkCudaErrors(hipMemset((void *)devPtrArowsIndex,0, sizeof(devPtrArowsIndex[0]) * matrixSizeArowsIndex));
    checkCudaErrors(hipMemset((void *)devPtrMval,      0, sizeof(devPtrMval[0])       * mSizeAval));

    memset(x,         0, arraySizeX           * sizeof(x[0]));
    //memset(f,         0, arraySizeF           * sizeof(f[0]));
    memset(r,         0, arraySizeR           * sizeof(r[0]));
    memset(rw,        0, arraySizeRW          * sizeof(rw[0]));
    memset(p,         0, arraySizeP           * sizeof(p[0]));
    memset(pw,        0, arraySizePW          * sizeof(pw[0]));
    memset(s,         0, arraySizeS           * sizeof(s[0]));
    memset(t,         0, arraySizeT           * sizeof(t[0]));
    memset(v,         0, arraySizeV           * sizeof(v[0]));

    /* create the test matrix and vectors on the host */
    checkCudaErrors(hipsparseSetMatType(descra,HIPSPARSE_MATRIX_TYPE_GENERAL));
    if (base) {
        checkCudaErrors(hipsparseSetMatIndexBase(descra,HIPSPARSE_INDEX_BASE_ONE));
        checkCudaErrors(hipsparseSetMatIndexBase(descrm,HIPSPARSE_INDEX_BASE_ONE));
    }
    else{
        checkCudaErrors(hipsparseSetMatIndexBase(descra,HIPSPARSE_INDEX_BASE_ZERO));
        checkCudaErrors(hipsparseSetMatIndexBase(descrm,HIPSPARSE_INDEX_BASE_ZERO));
    }
    checkCudaErrors(hipsparseSetMatType(descrm,HIPSPARSE_MATRIX_TYPE_GENERAL));

    for (int i=0; i<arraySizeX; i++) {
        x[i]=1.0; //x0
    }

    /* copy the csr matrix and vectors into device memory */


    checkCudaErrors(hipMemcpy (devPtrAval,       Aval,       (size_t)(matrixSizeAval       * sizeof(Aval[0])),       hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy (devPtrAcolsIndex, AcolsIndex, (size_t)(matrixSizeAcolsIndex * sizeof(AcolsIndex[0])), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy (devPtrArowsIndex, ArowsIndex, (size_t)(matrixSizeArowsIndex * sizeof(ArowsIndex[0])), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy (devPtrMval,       devPtrAval, (size_t)(matrixSizeAval       * sizeof(devPtrMval[0])), hipMemcpyDeviceToDevice));


    checkCudaErrors(hipMemcpy (devPtrX, x, (size_t)(arraySizeX * sizeof(devPtrX[0])), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy (devPtrF, f, (size_t)(arraySizeF * sizeof(devPtrF[0])), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy (devPtrR, r, (size_t)(arraySizeR * sizeof(devPtrR[0])), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy (devPtrRW,rw,(size_t)(arraySizeRW* sizeof(devPtrRW[0])),hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy (devPtrP, p, (size_t)(arraySizeP * sizeof(devPtrP[0])), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy (devPtrPW,pw,(size_t)(arraySizePW* sizeof(devPtrPW[0])),hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy (devPtrS, s, (size_t)(arraySizeS * sizeof(devPtrS[0])), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy (devPtrT, t, (size_t)(arraySizeT * sizeof(devPtrT[0])), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy (devPtrV, v, (size_t)(arraySizeV * sizeof(devPtrV[0])), hipMemcpyHostToDevice));

    /* --- GPU --- */
    /* create the analysis info (for lower and upper triangular factors) */
    checkCudaErrors(cusparseCreateSolveAnalysisInfo(&info_l));
    checkCudaErrors(cusparseCreateSolveAnalysisInfo(&info_u));

    /* analyse the lower and upper triangular factors */
    double ttl = second();
    checkCudaErrors(hipsparseSetMatFillMode(descrm,HIPSPARSE_FILL_MODE_LOWER));
    checkCudaErrors(hipsparseSetMatDiagType(descrm,HIPSPARSE_DIAG_TYPE_UNIT));
    checkCudaErrors(cusparseDcsrsv_analysis(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,matrixM,nnz,descrm,devPtrAval,devPtrArowsIndex,devPtrAcolsIndex,info_l));
    checkCudaErrors(hipDeviceSynchronize());
    double ttl2 = second();

    double ttu = second();
    checkCudaErrors(hipsparseSetMatFillMode(descrm,HIPSPARSE_FILL_MODE_UPPER));
    checkCudaErrors(hipsparseSetMatDiagType(descrm,HIPSPARSE_DIAG_TYPE_NON_UNIT));
    checkCudaErrors(cusparseDcsrsv_analysis(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,matrixM,nnz,descrm,devPtrAval,devPtrArowsIndex,devPtrAcolsIndex,info_u));
    checkCudaErrors(hipDeviceSynchronize());
    double ttu2= second();

    if(debug)
        printf("analysis lower %f (s), upper %f (s) \n",ttl2-ttl,ttu2-ttu);

    /* compute the lower and upper triangular factors using CUSPARSE csrilu0 routine (on the GPU) */
    double start_ilu, stop_ilu;
    if(debug)
        printf("CUSPARSE csrilu0 ");
    start_ilu= second();
    devPtrMrowsIndex = devPtrArowsIndex;
    devPtrMcolsIndex = devPtrAcolsIndex;
    checkCudaErrors(cusparseDcsrilu0(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,matrixM,descra,devPtrMval,devPtrArowsIndex,devPtrAcolsIndex,info_l));
    checkCudaErrors(hipDeviceSynchronize());
    stop_ilu = second();
    if(debug)
        fprintf (stdout, "time(s) = %10.8f \n",stop_ilu-start_ilu);

    auto t1 = second();
    gpu_pbicgstab(cublasHandle, cusparseHandle, matrixN, nnz,
                  descra, devPtrAval, devPtrArowsIndex, devPtrAcolsIndex,
                  descrm, devPtrMval, devPtrMrowsIndex, devPtrMcolsIndex,
                  info_l, info_u,
                  devPtrF,devPtrR,devPtrRW,devPtrP,devPtrPW,devPtrS,devPtrT,devPtrV,devPtrX, maxit, tol, debug);

    checkCudaErrors(hipDeviceSynchronize());
    auto t2 = second();
    *dtAlg = t2 - t1;

    /* destroy the analysis info (for lower and upper triangular factors) */
    checkCudaErrors(cusparseDestroySolveAnalysisInfo(info_l));
    checkCudaErrors(cusparseDestroySolveAnalysisInfo(info_u));

    /* copy the result into host memory */
    checkCudaErrors(hipMemcpy (res, devPtrX, (size_t)(arraySizeX * sizeof(double)), hipMemcpyDeviceToHost));


    free (x);
    free (r);
    free (rw);
    free (p);
    free (pw);
    free (s);
    free (t);
    free (v);
    if (devPtrX)    checkCudaErrors(hipFree (devPtrX));                    \
    if (devPtrF)    checkCudaErrors(hipFree (devPtrF));                    \
    if (devPtrR)    checkCudaErrors(hipFree (devPtrR));                    \
    if (devPtrRW)   checkCudaErrors(hipFree (devPtrRW));                   \
    if (devPtrP)    checkCudaErrors(hipFree (devPtrP));                    \
    if (devPtrS)    checkCudaErrors(hipFree (devPtrS));                    \
    if (devPtrT)    checkCudaErrors(hipFree (devPtrT));                    \
    if (devPtrV)    checkCudaErrors(hipFree (devPtrV));                    \
    if (devPtrAval) checkCudaErrors(hipFree (devPtrAval));                 \
    if (devPtrAcolsIndex) checkCudaErrors(hipFree (devPtrAcolsIndex));     \
    if (devPtrArowsIndex) checkCudaErrors(hipFree (devPtrArowsIndex));     \
    if (devPtrMval)       checkCudaErrors(hipFree (devPtrMval));           \
    if (cublasHandle)     checkCudaErrors(hipblasDestroy(cublasHandle));     \
    if (cusparseHandle)   checkCudaErrors(hipsparseDestroy(cusparseHandle)); \


    return true;
}



double rand_float_0_1() {
    double r = static_cast <double> (rand()) / static_cast <double> (RAND_MAX);

    return r;
}

double rand_float(double min, double max) {
    double norm = rand_float_0_1();

    return norm * (max - min) + min;
}

static bool gpu_pbicgstab2(
        hipblasHandle_t cublasHandle,
        hipsparseHandle_t cusparseHandle,
        int n,
        int nnz,
        const hipsparseMatDescr_t descrA,
	    double *A,
        int *iA,
        int *jA,
        double* x0,
        double* b,

        int maxit,
        double tol,
        bool debug,

        double* x, //out

        /*work, all zeroed*/
        double* r0,
        double* r,
        double* r_,
        double* v,
        double* v_,
        double* p,
        double* p_,
        double* s,
        double* t,
        double* h){


	//v = v_ = p = p_ = [0,0,0....0]
	double omega = 1;
	double alpha = 1;
	double beta = 0;
	double rho = 1;
	double rho_ = rho;

	double norm0;

	double one = 1;
	double mone = -1.0;
	double zero = 0;

	checkCudaErrors(hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &mone, descrA, A, iA, jA, x0, &zero, r));

   /* double r_host[3];
    hipMemcpy(r_host, r, 3 * sizeof(double), hipMemcpyDeviceToHost);
    std::ostringstream st;
    dump_vector(st, 3, r_host);
    std::cout << st.str() << std::endl;

    checkCudaErrors(hipblasDaxpy(cublasHandle, n, &one, b, 1, r, 1));
	checkCudaErrors(hipMemcpy(r0, r, sizeof(double) * n, hipMemcpyDeviceToDevice));*/


	checkCudaErrors(hipblasDnrm2(cublasHandle, n, r, 1, &norm0));


	if(debug){
        std::cout << "initial norm = " << norm0 << std::endl;
	}

	for(size_t i = 0; i < maxit; i++)
	{

        checkCudaErrors(hipblasDdot(cublasHandle, n, r0, 1, r, 1, &rho_));
        beta = (rho_ / rho) * (alpha / omega);
        double momega = -omega;
        checkCudaErrors(hipMemcpy(p_, v, sizeof(double) * n, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipblasDscal(cublasHandle, n, &momega, p_, 1));
        checkCudaErrors(hipblasDaxpy(cublasHandle,n, &one, p, 1, p_, 1));
        checkCudaErrors(hipblasDscal(cublasHandle, n, &beta, p_, 1));
        checkCudaErrors(hipblasDaxpy(cublasHandle,n, &one, r, 1, p_, 1));


		checkCudaErrors(hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, descrA, A, iA, jA, p_, &zero, v_));

       /* double *temp = static_cast<double *>(malloc(sizeof(double) * n));
        hipMemcpy(temp, v_, sizeof(double)*n, hipMemcpyDeviceToHost);
        std::cout << "temp" << std::endl;

        std::ostringstream ss;
        dump_vector(ss, n, temp);
        std::cout << ss.str() << std::endl;*/

		double dot_r_v;

		checkCudaErrors(hipblasDdot(cublasHandle, n, r0, 1, v_, 1, &dot_r_v));
		alpha = rho_ / dot_r_v;
		double malpha = -alpha;

		//std::cout << "alpha=" << alpha << std::endl;

        checkCudaErrors(hipMemcpy(h, p_, sizeof(double) * n, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipblasDscal(cublasHandle, n, &alpha, h, 1));
		checkCudaErrors(hipblasDaxpy(cublasHandle,n, &one, x0, 1, h, 1));

		checkCudaErrors(hipMemcpy(s, v_, sizeof(double) * n, hipMemcpyDeviceToDevice));
		checkCudaErrors(hipblasDscal(cublasHandle, n, &malpha, s, 1));
        checkCudaErrors(hipblasDaxpy(cublasHandle,n, &one, r, 1, s, 1));


		checkCudaErrors(hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, descrA, A, iA, jA, s, &zero, t));

		double num;
		double denum;
		checkCudaErrors(hipblasDdot(cublasHandle, n, t, 1, s, 1, &num));
		checkCudaErrors(hipblasDdot(cublasHandle, n, t, 1, t, 1, &denum));
		omega = num / denum;
		momega = -omega;


        checkCudaErrors(hipMemcpy(x, s, sizeof(double) * n, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipblasDscal(cublasHandle, n, &omega, x, 1));
		checkCudaErrors(hipblasDaxpy(cublasHandle,n, &one, h, 1, x, 1));

        checkCudaErrors(hipMemcpy(r_, t, sizeof(double) * n, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipblasDscal(cublasHandle, n, &momega, r_, 1));
		checkCudaErrors(hipblasDaxpy(cublasHandle,n, &one, s, 1, r_, 1));

		double norm;
		checkCudaErrors(hipblasDnrm2(cublasHandle, n, r_, 1, &norm));


        if(debug){
            std::cout << "k = " << i << ", norm = " << norm << std::endl;
        }

		if(norm < tol * norm0){
			return true;
		}


		if(abs(omega) < 1e-5 || isnan(omega)){
            if(debug){
                std::cout << "omega is close to zero, cannot continue" << std::endl;
                std::cout << "omega = " << omega << std::endl;
            }

            return false;
        }

		checkCudaErrors(hipMemcpy(r, r_, sizeof(double) * n, hipMemcpyDeviceToDevice));
		checkCudaErrors(hipMemcpy(p, p_, sizeof(double) * n, hipMemcpyDeviceToDevice));
		checkCudaErrors(hipMemcpy(v, v_, sizeof(double) * n, hipMemcpyDeviceToDevice));
		checkCudaErrors(hipMemcpy(x0, x, sizeof(double) * n, hipMemcpyDeviceToDevice));
		rho = rho_;
	}

	return false;


}

//TODO stupid code duplication
static bool gpu_pbicgstab2(
        hipblasHandle_t cublasHandle,
        hipsparseHandle_t cusparseHandle,
        int n,
        int nnz,
        const hipsparseMatDescr_t descrA,
        double *A0,
        int *iA0,
        int *jA0,
        double *d,
        double* x0,
        double* b,

        int maxit,
        double tol,
        bool debug,

        double* x, //out

        /*work, all zeroed*/
        double* r0,
        double* r,
        double* r_,
        double* v,
        double* v_,
        double* p,
        double* p_,
        double* s,
        double* t,
        double* h){


    //v = v_ = p = p_ = [0,0,0....0]
    double omega = 1;
    double alpha = 1;
    double beta = 0;
    double rho = 1;
    double rho_ = rho;

    double norm0;

    double one = 1;
    double mone = -1.0;
    double zero = 0;

    const int blockDim = 512;

    int blockCount;
    if(n % blockDim == 0){
        blockCount = n / blockDim;
    }else{
        blockCount = n / blockDim + 1;
    }





//    double r_host[3];
//    hipMemcpy(r_host, r, 3 * sizeof(double), hipMemcpyDeviceToHost);
//    std::ostringstream st;
//    dump_vector(st, 3, r_host);
//    std::cout << st.str() << std::endl;

    mult_spec<<<blockDim, blockCount>>>(n, x0, d, -1, r);
    checkCudaErrors(hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &mone, descrA, A0, iA0, jA0, x0, &one, r));


    checkCudaErrors(hipblasDaxpy(cublasHandle, n, &one, b, 1, r, 1));


    checkCudaErrors(hipMemcpy(r0, r, sizeof(double) * n, hipMemcpyDeviceToDevice));


    checkCudaErrors(hipblasDnrm2(cublasHandle, n, r, 1, &norm0));


    if(debug){
        std::cout << "initial norm = " << norm0 << std::endl;
    }

    for(size_t i = 0; i < maxit; i++)
    {

        checkCudaErrors(hipblasDdot(cublasHandle, n, r0, 1, r, 1, &rho_));
        beta = (rho_ / rho) * (alpha / omega);
        double momega = -omega;
        checkCudaErrors(hipMemcpy(p_, v, sizeof(double) * n, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipblasDscal(cublasHandle, n, &momega, p_, 1));
        checkCudaErrors(hipblasDaxpy(cublasHandle,n, &one, p, 1, p_, 1));
        checkCudaErrors(hipblasDscal(cublasHandle, n, &beta, p_, 1));
        checkCudaErrors(hipblasDaxpy(cublasHandle,n, &one, r, 1, p_, 1));


        mult_spec<<<blockDim, blockCount>>>(n, p_, d, 1, v_);
        checkCudaErrors(hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, descrA, A0, iA0, jA0, p_, &one, v_));

        /* double *temp = static_cast<double *>(malloc(sizeof(double) * n));
         hipMemcpy(temp, v_, sizeof(double)*n, hipMemcpyDeviceToHost);
         std::cout << "temp" << std::endl;

         std::ostringstream ss;
         dump_vector(ss, n, temp);
         std::cout << ss.str() << std::endl;*/

        double dot_r_v;

        checkCudaErrors(hipblasDdot(cublasHandle, n, r0, 1, v_, 1, &dot_r_v));
        alpha = rho_ / dot_r_v;
        double malpha = -alpha;

        //std::cout << "alpha=" << alpha << std::endl;

        checkCudaErrors(hipMemcpy(h, p_, sizeof(double) * n, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipblasDscal(cublasHandle, n, &alpha, h, 1));
        checkCudaErrors(hipblasDaxpy(cublasHandle,n, &one, x0, 1, h, 1));

        checkCudaErrors(hipMemcpy(s, v_, sizeof(double) * n, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipblasDscal(cublasHandle, n, &malpha, s, 1));
        checkCudaErrors(hipblasDaxpy(cublasHandle,n, &one, r, 1, s, 1));


        mult_spec<<<blockDim, blockCount>>>(n, s, d, 1, t);
        checkCudaErrors(hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, descrA, A0, iA0, jA0, s, &one, t));

        double num;
        double denum;
        checkCudaErrors(hipblasDdot(cublasHandle, n, t, 1, s, 1, &num));
        checkCudaErrors(hipblasDdot(cublasHandle, n, t, 1, t, 1, &denum));
        omega = num / denum;
        momega = -omega;


        checkCudaErrors(hipMemcpy(x, s, sizeof(double) * n, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipblasDscal(cublasHandle, n, &omega, x, 1));
        checkCudaErrors(hipblasDaxpy(cublasHandle,n, &one, h, 1, x, 1));

        checkCudaErrors(hipMemcpy(r_, t, sizeof(double) * n, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipblasDscal(cublasHandle, n, &momega, r_, 1));
        checkCudaErrors(hipblasDaxpy(cublasHandle,n, &one, s, 1, r_, 1));

        double norm;
        checkCudaErrors(hipblasDnrm2(cublasHandle, n, r_, 1, &norm));


        if(debug){
            std::cout << "k = " << i << ", norm = " << norm << std::endl;
        }

        if(norm < tol * norm0){
            return true;
        }


        if(abs(omega) < 1e-5 || isnan(omega)){
            if(debug){
                std::cout << "omega is close to zero, cannot continue" << std::endl;
                std::cout << "omega = " << omega << std::endl;
            }

            return false;
        }

        checkCudaErrors(hipMemcpy(r, r_, sizeof(double) * n, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipMemcpy(p, p_, sizeof(double) * n, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipMemcpy(v, v_, sizeof(double) * n, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipMemcpy(x0, x, sizeof(double) * n, hipMemcpyDeviceToDevice));
        rho = rho_;
    }

    return false;


}

bool bicgstab(int n, int nnz, double *A, int *iA, int *jA, double *b, int maxit, double tol, bool debug, double *x, double *dtAlg){


    hipblasHandle_t cublasHandle  = nullptr;
    hipsparseHandle_t cusparseHandle  = nullptr;
    hipsparseMatDescr_t descrA= nullptr;
    hipsparseStatus_t status1;
    double *dev_A       = nullptr;
    int    *dev_iA = nullptr;
    int    *dev_jA = nullptr;
    double *dev_x0 = nullptr;
    double *dev_b = nullptr;
    double *dev_x = nullptr;
    double *dev_r0 = nullptr;
    double *dev_r = nullptr;
    double *dev_r_ = nullptr;
    double *dev_v = nullptr;
    double *dev_v_ = nullptr;
    double *dev_p = nullptr;
    double *dev_p_ = nullptr;
    double *dev_s = nullptr;
    double *dev_t = nullptr;
    double *dev_h = nullptr;



    int base = iA[0];

    /* initialize cublas */
    if (hipblasCreate(&cublasHandle) != HIPBLAS_STATUS_SUCCESS) {
        fprintf( stderr, "!!!! CUBLAS initialization error\n" );
        return false;
    }
    /* initialize cusparse */
    status1 = hipsparseCreate(&cusparseHandle);
    if (status1 != HIPSPARSE_STATUS_SUCCESS) {
        fprintf( stderr, "!!!! CUSPARSE initialization error\n" );
        return false;
    }
    /* create three matrix descriptors */
    status1 = hipsparseCreateMatDescr(&descrA);
    if ((status1 != HIPSPARSE_STATUS_SUCCESS)){
        fprintf( stderr, "!!!! CUSPARSE hipsparseCreateMatDescr (coefficient matrix) error\n" );
        return false;
    }

    /* allocate device memory for csr matrix and vectors */
    checkCudaErrors(hipMalloc ((void**)&dev_A,  sizeof(double) * nnz));
    checkCudaErrors(hipMalloc ((void**)&dev_iA, sizeof(int) * (n + 1)));
    checkCudaErrors(hipMalloc ((void**)&dev_jA, sizeof(int) * nnz));
    checkCudaErrors(hipMalloc ((void**)&dev_x0, sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_b,  sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_x,  sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_r0, sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_r,  sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_r_, sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_v,  sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_v_, sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_p,  sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_p_, sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_s,  sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_t,  sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_h,  sizeof(double) * n));


    checkCudaErrors(hipMemcpy(dev_A, A, sizeof(double) * nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_iA, iA, sizeof(int) * (n + 1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_jA, jA, sizeof(int) * nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_b, b, sizeof(double) * n, hipMemcpyHostToDevice));


    double *x0 = new double[n]; //x0 = [1,1,1,...1]
    for (int i = 0; i < n; ++i) {
        x0[i] = 1;
    }
    checkCudaErrors(hipMemcpy(dev_x0, x0, sizeof(double) * n, hipMemcpyHostToDevice));
    delete[] x0;


    //checkCudaErrors(hipMemset((void *)dev_x0,          0, sizeof(double)* n)); //x0 is zero vector
    checkCudaErrors(hipMemset((void *)dev_x,           0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_r0,          0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_r,           0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_r_,          0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_v,           0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_v_,          0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_p,           0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_p_,          0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_s,           0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_t,           0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_h,           0, sizeof(double)* n));



    /* create the test matrix and vectors on the host */
    checkCudaErrors(hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL));
    if (base) {
        checkCudaErrors(hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ONE));
    }
    else{
        checkCudaErrors(hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO));
    }


    auto t1 = second();
    auto res = gpu_pbicgstab2(cublasHandle,
                              cusparseHandle,
                              n,
                              nnz,
                              descrA,
                              dev_A,
                              dev_iA,
                              dev_jA,
                              dev_x0,
                              dev_b,
                              maxit,
                              tol,
                              debug,
                              dev_x,
                              dev_r0,
                              dev_r,
                              dev_r_,
                              dev_v,
                              dev_v_,
                              dev_p,
                              dev_p_,
                              dev_s,
                              dev_t,
                              dev_h
    );
    auto t2 = second();
    *dtAlg = t2 - t1;

    checkCudaErrors(hipDeviceSynchronize());


    /* copy the result into host memory */
    checkCudaErrors(hipMemcpy (x, dev_x, sizeof(double) * n, hipMemcpyDeviceToHost));


    hipsparseDestroyMatDescr(descrA);
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);



    checkCudaErrors(hipFree(dev_A));
    checkCudaErrors(hipFree(dev_iA));
    checkCudaErrors(hipFree(dev_jA));
    checkCudaErrors(hipFree(dev_x0));
    checkCudaErrors(hipFree(dev_b));
    checkCudaErrors(hipFree(dev_x));
    checkCudaErrors(hipFree(dev_r0));
    checkCudaErrors(hipFree(dev_r));
    checkCudaErrors(hipFree(dev_r_));
    checkCudaErrors(hipFree(dev_v));
    checkCudaErrors(hipFree(dev_v_));
    checkCudaErrors(hipFree(dev_p));
    checkCudaErrors(hipFree(dev_p_));
    checkCudaErrors(hipFree(dev_s));
    checkCudaErrors(hipFree(dev_t));
    checkCudaErrors(hipFree(dev_h));



    return res;
}


//TODO another stupid duplication
bool bicgstab(int n, int nnz, double *A0, int *iA0, int *jA0, double *d, double *x0, double *b, int maxit, double tol, bool debug, double *x, double *dtAlg){


    hipblasHandle_t cublasHandle  = nullptr;
    hipsparseHandle_t cusparseHandle  = nullptr;
    hipsparseMatDescr_t descrA= nullptr;
    hipsparseStatus_t status1;
    double *dev_A       = nullptr;
    int    *dev_iA = nullptr;
    int    *dev_jA = nullptr;
    double *dev_x0 = nullptr;
    double *dev_b = nullptr;
    double *dev_x = nullptr;
    double *dev_r0 = nullptr;
    double *dev_r = nullptr;
    double *dev_r_ = nullptr;
    double *dev_v = nullptr;
    double *dev_v_ = nullptr;
    double *dev_p = nullptr;
    double *dev_p_ = nullptr;
    double *dev_s = nullptr;
    double *dev_t = nullptr;
    double *dev_h = nullptr;
    double *dev_d = nullptr;



    int base = iA0[0];

    /* initialize cublas */
    if (hipblasCreate(&cublasHandle) != HIPBLAS_STATUS_SUCCESS) {
        fprintf( stderr, "!!!! CUBLAS initialization error\n" );
        return false;
    }
    /* initialize cusparse */
    status1 = hipsparseCreate(&cusparseHandle);
    if (status1 != HIPSPARSE_STATUS_SUCCESS) {
        fprintf( stderr, "!!!! CUSPARSE initialization error\n" );
        return false;
    }
    /* create three matrix descriptors */
    status1 = hipsparseCreateMatDescr(&descrA);
    if ((status1 != HIPSPARSE_STATUS_SUCCESS)){
        fprintf( stderr, "!!!! CUSPARSE hipsparseCreateMatDescr (coefficient matrix) error\n" );
        return false;
    }

    /* allocate device memory for csr matrix and vectors */
    checkCudaErrors(hipMalloc ((void**)&dev_A,  sizeof(double) * nnz));
    checkCudaErrors(hipMalloc ((void**)&dev_iA, sizeof(int) * (n + 1)));
    checkCudaErrors(hipMalloc ((void**)&dev_jA, sizeof(int) * nnz));
    checkCudaErrors(hipMalloc ((void**)&dev_x0, sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_b,  sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_x,  sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_r0, sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_r,  sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_r_, sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_v,  sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_v_, sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_p,  sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_p_, sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_s,  sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_t,  sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_h,  sizeof(double) * n));
    checkCudaErrors(hipMalloc ((void**)&dev_d,  sizeof(double) * n));

    checkCudaErrors(hipMemcpy(dev_A, A0, sizeof(double) * nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_iA, iA0, sizeof(int) * (n + 1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_jA, jA0, sizeof(int) * nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_b, b, sizeof(double) * n, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_d, d, sizeof(double) * n, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_x0, x0, sizeof(double) * n, hipMemcpyHostToDevice));


    //checkCudaErrors(hipMemset((void *)dev_x0,          0, sizeof(double)* n)); //x0 is zero vector
    checkCudaErrors(hipMemset((void *)dev_x,           0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_r0,          0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_r,           0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_r_,          0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_v,           0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_v_,          0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_p,           0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_p_,          0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_s,           0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_t,           0, sizeof(double)* n));
    checkCudaErrors(hipMemset((void *)dev_h,           0, sizeof(double)* n));



    /* create the test matrix and vectors on the host */
    checkCudaErrors(hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL));
    if (base) {
        checkCudaErrors(hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ONE));
    }
    else{
        checkCudaErrors(hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO));
    }


    auto t1 = second();
    auto res = gpu_pbicgstab2(cublasHandle,
                              cusparseHandle,
                              n,
                              nnz,
                              descrA,
                              dev_A,
                              dev_iA,
                              dev_jA,
                              dev_d,
                              dev_x0,
                              dev_b,
                              maxit,
                              tol,
                              debug,
                              dev_x,
                              dev_r0,
                              dev_r,
                              dev_r_,
                              dev_v,
                              dev_v_,
                              dev_p,
                              dev_p_,
                              dev_s,
                              dev_t,
                              dev_h
    );
    auto t2 = second();
    *dtAlg = t2 - t1;

    checkCudaErrors(hipDeviceSynchronize());


    /* copy the result into host memory */
    checkCudaErrors(hipMemcpy (x, dev_x, sizeof(double) * n, hipMemcpyDeviceToHost));


    hipsparseDestroyMatDescr(descrA);
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);



    checkCudaErrors(hipFree(dev_A));
    checkCudaErrors(hipFree(dev_iA));
    checkCudaErrors(hipFree(dev_jA));
    checkCudaErrors(hipFree(dev_x0));
    checkCudaErrors(hipFree(dev_b));
    checkCudaErrors(hipFree(dev_x));
    checkCudaErrors(hipFree(dev_r0));
    checkCudaErrors(hipFree(dev_r));
    checkCudaErrors(hipFree(dev_r_));
    checkCudaErrors(hipFree(dev_v));
    checkCudaErrors(hipFree(dev_v_));
    checkCudaErrors(hipFree(dev_p));
    checkCudaErrors(hipFree(dev_p_));
    checkCudaErrors(hipFree(dev_s));
    checkCudaErrors(hipFree(dev_t));
    checkCudaErrors(hipFree(dev_h));



    return res;
}




void gen_rand_vector(int n, double *vector, double probability_of_zero, double min, double max) {
	for (int i = 0; i < n; ++i) {
		vector[i] = rand_float_0_1() <= probability_of_zero ? 0.0 : rand_float(min, max);
	}
}



void toDenseVector(int n, int nnz, double* A, int* IA, double* out) {
    int sum = IA[0]; //base
    int count = 0;
    for (int i = 0; i < n; ++i) {
        if (IA[i + 1] - sum > 0) {
            out[i] = A[count++];
            sum = IA[i + 1];

        }
        else {
            out[i] = 0.0;
        }

    }
}






